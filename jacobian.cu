#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include "f_eval.cuh"

__global__ void kernelCompute_shared(double h, int N, int M, double* d_data, double* d_out) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    extern __shared__ double shared_data[];
    shared_data[threadIdx.x] = d_data[index];
    __syncthreads();

    if (index < N * M) {
        int which_m = threadIdx.x / M; 
        int position = threadIdx.x - which_m * M;

        double* temp_array = (double*)malloc(sizeof(double) * M);
        memcpy(temp_array, shared_data + which_m * M, M * sizeof(double));

        double temp_minus = temp_array[position] - h;
        double temp_plus = temp_array[position] + h;

        temp_array[position] = temp_minus;
        double output_minus = f_eval(temp_array, M);

        temp_array[position] = temp_plus;
        double output_plus = f_eval(temp_array, M);

        free(temp_array);

        double output = (output_plus - output_minus) / (2 * h);

        d_out[index] = output;
    }
}

__global__ void kernelCompute(double h, int N, int M, double* d_data, double* d_out) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < N * M) {
        double* temp_array = (double*)malloc(sizeof(double) * M);
        memcpy(temp_array, d_data + (index / M * M), M * sizeof(double));

        double temp_minus = temp_array[index - index / M * M] - h;
        double temp_plus = temp_array[index - index / M * M] + h;

        temp_array[index - index / M * M] = temp_minus;
        double output_minus = f_eval(temp_array, M);

        temp_array[index - index / M * M] = temp_plus;
        double output_plus = f_eval(temp_array, M);

        double output = (output_plus - output_minus) / (2 * h);

        d_out[index] = output;

        free(temp_array);

        /* Old verison

        double* temp_minus = (double*)malloc(sizeof(double) * M);
        memcpy(temp_minus, d_data + (index / M * M), M * sizeof(double));
        double t_minus = temp_minus[index - index / M * M];
        temp_minus[index - index / M * M] = t_minus - h;

        double* temp_plus = (double*)malloc(sizeof(double) * M);
        memcpy(temp_plus, d_data + (index / M * M), M * sizeof(double));
        double t_plus = temp_plus[index - index / M * M];
        temp_plus[index - index / M * M] = t_plus + h;

        double output = (f_eval(temp_plus, M) - f_eval(temp_minus, M));

        // printf("%d, %f, %f\n", index - index / M * M, temp_plus[index - index / M * M], temp_minus[index - index / M * M]);
        // printf("%d, %f\n", index, f_eval(temp_plus, M) - f_eval(temp_minus, M));

        d_out[index] = output / (2 * h);
        
        */
    }
}

int main(int argc, char *argv[]) {
    if(argc != 4){
        printf("Input is not correct!\n");
        exit(1);
    }

    FILE* input = fopen(argv[1], "r");
    if (input == NULL) {
        perror("Error: Read File Error");
        exit(1);
    }

    int N = 0;
    int M = 0;

    fscanf(input, "%d", &N);
    fscanf(input, "%d", &M);

    //printf("%d, %d\n", N, M);

    /* 2D array
    double** data = (double**)malloc(N * sizeof(double*));
    for (i = 0; i < N; i++) {
        data[i] = (double*)malloc(M * sizeof(double));
    }

    for(i = 0; i < N; i++){
        for(j = 0; j < M ; j++){
            double temp = 0.0f;
            fscanf(input, "%lf,", &temp);
            data[i][j] = (double)temp;

            //debug info
            //printf("%lf ", data[i][j]);
        }

        //debug info
        //printf("\n");
    }

    //printf("%f\n", f_eval(data[i], M));
    */

    double* data = (double*)malloc(N * M * sizeof(double));
    for(int i = 0; i < N * M; i++){
        double temp = 0.0f;
        fscanf(input, "%lf,", &temp);
        data[i] = (double)temp;
    }

    fclose(input);
    
    FILE* output = fopen(argv[2], "w");
    double h = (double)atof(argv[3]);
    // printf("h on host is: %f\n", h);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    double* d_data;
    hipMalloc((void**)&d_data, sizeof(double) * N * M);
    
    double* d_out;
    hipMalloc((void**)&d_out, sizeof(double) * N * M);

    hipMemcpy(d_data, data, sizeof(double) * N * M, hipMemcpyHostToDevice);

    hipEventRecord(start, 0);

    if (M < 1024) {
        int block_size = 1024 / M * M;
        kernelCompute_shared<<<(N * M + block_size - 1) / block_size, block_size, block_size * sizeof(double)>>>(h, N, M, d_data, d_out);
    }
    else {
        kernelCompute<<<(N * M + 1023) / 1024, 1024>>>(h, N, M, d_data, d_out);
    }

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop); 

    double* out = (double*)malloc(N * M * sizeof(double));
    hipMemcpy(out, d_out, sizeof(double) * N * M, hipMemcpyDeviceToHost);

    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);

    printf("Time spent: %f\n", elapsedTime);

    for(int i = 0; i < N * M; i++){
        if ((i + 1) % M == 0){
            fprintf(output, "%f\n", out[i]);
        }
        else {
            fprintf(output, "%f ", out[i]);
        }
    }

    free(data);
    free(out);
    hipFree(d_data);
    hipFree(d_out);

    // FILE* input_long = fopen("input_long", "w");
    // for(int i = 0; i < 1000; i++) {
    //     fprintf(input_long, "p_x[%d] + ", i);
    // }

    return 0;
}
